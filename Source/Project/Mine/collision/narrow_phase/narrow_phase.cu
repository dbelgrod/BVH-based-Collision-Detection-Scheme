#include "hip/hip_runtime.h"
#include "narrow_phase.cuh"
#include "collision/lbvh/BvhBV.h"
#include "utility/CudaDeviceUtils.h"
#include <gProximity/cuda_intersect_tritri.h>

namespace mn {

	__global__ void simpleNarrowPhase(uint numpair, int2* _cps, const int3* _indices, const PointType* _vertices, int* _actualCpNum) {
		int	idx = blockIdx.x * blockDim.x + threadIdx.x;
		if (idx >= numpair) return;
		const int2 cp = _cps[idx];
		const PointType P0 = _vertices[_indices[cp.x].x];
		const PointType P1 = _vertices[_indices[cp.x].y];
		const PointType P2 = _vertices[_indices[cp.x].z];
		const PointType Q0 = _vertices[_indices[cp.y].x];
		const PointType Q1 = _vertices[_indices[cp.y].y];
		const PointType Q2 = _vertices[_indices[cp.y].z];
		if (!triangleIntersection2(make_float3(P0.x, P0.y, P0.z), make_float3(P1.x, P1.y, P1.z), make_float3(P2.x, P2.y, P2.z),
			make_float3(Q0.x, Q0.y, Q0.z), make_float3(Q1.x, Q1.y, Q1.z), make_float3(Q2.x, Q2.y, Q2.z))) {
			_cps[idx] = { -1, -1 };
		}
		else
			atomicAggInc(_actualCpNum);
	}
}
